#include "hip/hip_runtime.h"
/*
M: sparse adjacency matrix, 
get(M, i, j) = the link from 'j' to 'i', such that for all 'j':
   sum(i, get(M, i, j)) = 1

d: damping factor

v_quadratic_error: quadratic error allowed

returns a dense vector of ranks
*/

#include "SparseMatrix.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "hipsparse.h"

#define QUADRATIC_ERROR .001
#define DAMPING_FACTOR .85

using namespace std;


/*
float * pagerank(SparseMatrix M) {
   int n = M.numRows();
   float *v = malloc(sizeof(float) * n);
   std::fill(v, v+n, 1/n);

   float *last_v = malloc(sizeof(float) * n);

   M_hat = sparse_MXplusB(M, DAMPING_FACTOR, (1-DAMPING_FACTOR)/n);

   float error;
   do {
      last_v = v;
      v = matrixVectorMultiply(M_hat, v);

      error = vectorSubtractAndNormalize2(v, last_v, n);
      //    = sum( (v[i] - last_v[i])^2 ) ^ .5 = norm(v - last_v, 2)

   } while (error > QUADRATIC_ERROR);


   free(v);
   free(M_hat);
   free(last_v);
}
*/

void convertCOO2CSR(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowPtr;

   //hipsparseStatus_t status;
   hipMalloc(&devRowPtr, M->nnz * sizeof(int));

   hipsparseXcoo2csr(handle, M->devRowInd, M->nnz, M->width, devRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
   //status = hipsparseXcoo2csr(handle, M->devRowInd, M->nnz, M->width, devRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
   M->devRowPtr = devRowPtr;

   hipFree(M->devRowInd);
}

void putMatOnDevice(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowInd, *devColInd;
   float *devVal;
   hipMalloc(&devRowInd, M->nnz * sizeof(int));
   hipMemcpy(devRowInd, M->cooRowIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 
   hipMalloc(&devVal, M->nnz * sizeof(float));
   hipMemcpy(devVal, M->cooValA, M->nnz * sizeof(float), hipMemcpyHostToDevice);
   hipMalloc(&devColInd, M->nnz * sizeof(int));
   hipMemcpy(devColInd, M->cooColIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 

   M->devRowInd = devRowInd;
   M->devVal = devVal;
   M->devColInd = devColInd;
}

/* TODO

float *sparse_MatrixVectorMultiply(SparseMatrix M, float *v, int n)

SparseMatrix *sparse_MXplusB(SparseMatrix X, float m, float b, int n)

float vectorSubtractAndNormalize2(float *v, float *last_v, int n)


*/
//TODO test coo2csr on test data
//TODO test csrmv on test data
void sparse_MatrixVectorMultiply(SparseMatrix *M, hipsparseHandle_t handle, float *vect, float *newVect, float **devVect) {
#ifdef GPU
  // hipsparseStatus_t hipsparseXcoo2csr(hipsparseHandle_t handle, const int *cooRowInd,
   //                          int nnz, int m, int *csrRowPtr, hipsparseIndexBase_t idxBase);
   float *devVectNew;
   float alpha = 1.0f;
   float beta = 0.0f;
   int vectWidth = M->width;

   //hipsparseStatus_t status;
   hipsparseMatDescr_t descr;
   //status = hipsparseCreateMatDescr(&descr);
   hipsparseCreateMatDescr(&descr);
   hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

   // Check to see if the prestige vector is already on card
   if ( *devVect == NULL ) {
      hipMalloc(devVect, vectWidth * sizeof(float));
      hipMemcpy(*devVect, vect, vectWidth * sizeof(float), hipMemcpyHostToDevice);
   } 

   hipMalloc(&devVectNew, vectWidth * sizeof(float));

   hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
   hipsparseScsrmv(handle, op, M->width, M->width, M->nnz, &alpha,
         descr, M->devVal, M->devRowPtr, M->devColInd, *devVect, &beta, devVectNew); 

   hipMemcpy(newVect, devVectNew, vectWidth * sizeof(float), hipMemcpyDeviceToHost);
   *devVect = devVectNew;

   hipFree(*devVect);
#endif
}

__global__ void MXplusBKernel(int n, float m, float *x, float b) {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) {
      x[i] = m * x[i] + b;
   }
}

void sparse_MXplusB(SparseMatrix *M, hipsparseHandle_t handle, float m, float b) {
   int numBlocks = (M->nnz + 1023) / 1024;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(1024, 1);

   MXplusBKernel<<<gridDim, blockDim>>>(M->nnz, m, M->devVal, b);
}


int main() {
   float vals[7] = {1.0, 4.0, 2.0, 3.0, 5.0, 7.0, 9.0};
   int rowInd[7] = {0, 0, 1, 1, 2, 2, 3};
   int colInd[7] = {0, 1, 1, 2, 0, 3, 2};

   int n = 4;
   float vect[4] = {1.0, 2.0, 3.0, 4.0};
   float newVect[4];
   float *devVect = NULL;

   hipsparseHandle_t handle;
   //hipsparseStatus_t status;
   hipsparseCreate(&handle);
   //status = hipsparseCreate(&handle);
   SparseMatrix m (vals, rowInd, colInd, n, 7);

   for (int i = 0; i < n; i++) {
      printf("%.2f ", vect[i]);
   }
   printf("\n");

   putMatOnDevice(&m, handle);
   sparse_MXplusB(&m, handle, DAMPING_FACTOR, (1.0f - DAMPING_FACTOR) / n);
   convertCOO2CSR(&m, handle);

   //float error = 1000000000000.0f;

   // while (error > QUADRATICERROR) {
      sparse_MatrixVectorMultiply(&m, handle,  vect, newVect, &devVect);
      //Calculate error

      //swap pointers
      /*
      float *temp = newVect;
      newVect = vect;
      vect = temp;
      */
   // }
      
   for (int i = 0; i < n; i++) {
      printf("%.2f ", newVect[i]);
      // SHOULD BE [9 13 33 27]
   }
   printf("\n");
   return 0;
}
