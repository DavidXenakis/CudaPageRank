#include "hip/hip_runtime.h"
/*
M: sparse adjacency matrix, 
get(M, i, j) = the link from 'j' to 'i', such that for all 'j':
   sum(i, get(M, i, j)) = 1

d: damping factor

v_quadratic_error: quadratic error allowed

returns a dense vector of ranks
*/

#include "SparseMatrix.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include "hipsparse.h"

#define QUADRATIC_ERROR .001
#define DAMPING_FACTOR .85
#define THREADS_PER_BLOCK 1024
#ifdef MIC
float vectorSubtractAndNormalize2(float *v, float *last_v, int n) {
   float sum = 0;

   #pragma omp parallel for reduction(+:sum)
   for (int i = 0; i < n; i++) {
      sum = sum + (v[i] - last_v[i]) * (v[i] - last_v[i]);
   }
}
#endif

#ifdef GPU
__global__ void subtractAndSquare(float *vectNew, float *vect, float *dest, int *n) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   float val;
   if (idx < n[0]){
      val = (vectNew[idx] - vect[idx]);// * (vectNew[idx] - vect[idx]);
      /*
      dest[idx] = val * val;
      */
      dest[idx] = val < 0 ? val * -1.0 : val;
   }
}
#endif

#ifdef GPU
float vectorSubtractAndNormalize2(float *devVectNew, float *devVect, int n) {
   //to be parallelized
   float sum = 0;

   //allocate on gpu
   float *d_difference;
   hipMalloc(&d_difference, n * sizeof(float));
   int *d_n;
   hipMalloc(&d_n, sizeof(int));

   //copy to gpu
   hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

   int numBlocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   //launch kenerl
   subtractAndSquare <<<gridDim, blockDim>>> (devVectNew, devVect, d_difference, d_n);

   //copy result from gpu
   float *difference = (float *) malloc(n * sizeof(float));
   hipMemcpy(difference, d_difference, n * sizeof(float), hipMemcpyDeviceToHost);

   //free from gpu
   hipFree(d_difference);
   hipFree(d_n);

   sum = thrust::reduce(difference, difference + n);
/*
   #else //neither mic nor gpu
   for (int i = 0; i < n; i++) {
      sum += (v[i] - last_v[i]) * (v[i] - last_v[i]);
   }
   #endif
*/
   //return sqrtf(sum);
   return sum;
}
#endif

void convertCOO2CSR(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowPtr;

   //hipsparseStatus_t status;
   hipMalloc(&devRowPtr, M->nnz * sizeof(int));

   hipsparseXcoo2csr(handle, M->devRowInd, M->nnz, M->width, devRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
   M->devRowPtr = devRowPtr;

   hipFree(M->devRowInd);
}

void putMatOnDevice(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowInd, *devColInd;
   float *devVal;
   hipMalloc(&devRowInd, M->nnz * sizeof(int));
   hipMemcpy(devRowInd, M->cooRowIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 
   hipMalloc(&devVal, M->nnz * sizeof(float));
   hipMemcpy(devVal, M->cooValA, M->nnz * sizeof(float), hipMemcpyHostToDevice);
   hipMalloc(&devColInd, M->nnz * sizeof(int));
   hipMemcpy(devColInd, M->cooColIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 

   M->devRowInd = devRowInd;
   M->devVal = devVal;
   M->devColInd = devColInd;
}

#ifdef GPU
void sparse_MatrixVectorMultiply(SparseMatrix *M, hipsparseHandle_t handle, float *vect, float *newVect, float **devVect, float **devVectNew) {
   float alpha = 1.0f;
   float beta = 0.0f;
   int vectWidth = M->width;

   hipsparseMatDescr_t descr;
   hipsparseCreateMatDescr(&descr);
   hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

   // Check to see if the prestige vector is already on card
   if ( *devVect == NULL ) {
      hipMalloc(devVect, vectWidth * sizeof(float));
      hipMemcpy(*devVect, vect, vectWidth * sizeof(float), hipMemcpyHostToDevice);
   } 
   if ( *devVectNew == NULL ) {
      hipMalloc(devVectNew, vectWidth * sizeof(float));
   }

   hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
   hipsparseScsrmv(handle, op, M->width, M->width, M->nnz, &alpha,
         descr, M->devVal, M->devRowPtr, M->devColInd, *devVect, &beta, *devVectNew); 

   hipMemcpy(newVect, *devVectNew, vectWidth * sizeof(float), hipMemcpyDeviceToHost);
}
#endif

__global__ void MXKernel(int n, float m, float *x) {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) {
      x[i] = m * x[i];
   }
}

__global__ void XplusBKernel(int n, float *x, float b) {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) {
      x[i] = x[i] + b;
   }
}

void sparse_MX(SparseMatrix *M, float m) {
   int numBlocks = (M->nnz + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   MXKernel<<<gridDim, blockDim>>>(M->nnz, m, M->devVal);
}

void sparse_XplusB(float *devVect, int width, float b) {
   int numBlocks = (width + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   XplusBKernel<<<gridDim, blockDim>>>(width, devVect, b);
}

void pageRank(SparseMatrix *M, int *array) {
   int n = M->width;
   float *vect = (float *) malloc(sizeof(float) * n);
   std::fill(vect, vect + n, 1.0f / n);

   float *newVect = (float *) malloc(sizeof(float) * n);
   float *devVect = NULL;
   float *devVectNew = NULL;

   hipsparseHandle_t handle;
   hipsparseCreate(&handle);

   putMatOnDevice(M, handle);
   sparse_MX(M, DAMPING_FACTOR);
   convertCOO2CSR(M, handle);

   float b = (1.0f - DAMPING_FACTOR) / n;
   printf("b = %f\n", b);
   int iter = 0;

   float error;
   do {
      sparse_MatrixVectorMultiply(M, handle, vect, newVect, &devVect, &devVectNew);

      sparse_XplusB(devVectNew, n, b);

      error = vectorSubtractAndNormalize2(devVectNew, devVect, n);

      // Swap old and new vectors to reuse space
      float *temp = newVect;
      newVect = vect;
      vect = temp;

      temp = devVectNew;
      devVectNew = devVect;
      devVect = temp;
      printf("Iteration: %d... Error: %.4f\n", iter++, error);
      
   } while (error > QUADRATIC_ERROR);

   hipMemcpy(vect, devVect, n * sizeof(float), hipMemcpyDeviceToHost);

   hipFree(devVectNew);
   hipFree(devVect);
   hipFree(M->devVal);
   hipFree(M->devRowPtr);
   hipFree(M->devColInd);
   
   free(newVect);
   for(int i = 0; i < M->width; i++) {
      array[i] = i;
   }

   thrust::stable_sort_by_key(vect, vect + n, array, thrust::greater<float>());

   M->sortedPrestigeVector = vect;
}
/*
int main() {
   float vals[7] = {.5, .5, .5, .5, .5, 1, .5};
   int rowInd[7] = {0, 0, 1, 1, 2, 2, 3};
   int colInd[7] = {0, 1, 1, 2, 0, 3, 2};

   float vals2[9] = {1.0, .25, 1, 1, .5, .25, .25, .25, .5};
   int rowInd2[9] = {0, 1, 1, 1, 1, 2, 3, 4, 4};
   int colInd2[9] = {1, 0, 2, 3, 4, 0, 0, 0, 4};

   SparseMatrix m (vals, rowInd, colInd, 4, 7);
   SparseMatrix m2(vals2, rowInd2, colInd2, 5, 9);

   pageRank(m);
   pageRank(m2);
     
   return 0;
}*/
