#include "hip/hip_runtime.h"
/*
M: sparse adjacency matrix, 
get(M, i, j) = the link from 'j' to 'i', such that for all 'j':
   sum(i, get(M, i, j)) = 1

d: damping factor

v_quadratic_error: quadratic error allowed

returns a dense vector of ranks
*/

#include "SparseMatrix.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include "hipsparse.h"

#define QUADRATIC_ERROR .0001
#define DAMPING_FACTOR .85
#define THREADS_PER_BLOCK 1024
#ifdef MIC
float vectorSubtractAndNormalize2(float *v, float *last_v, int n) {
   float sum = 0;

   #pragma omp parallel for reduction(+:sum)
   for (int i = 0; i < n; i++) {
      sum = sum + (v[i] - last_v[i]) * (v[i] - last_v[i]);
   }
}
#endif

#ifdef GPU
__global__ void subtractAndSquare(float *vectNew, float *vect, float *dest, int *n) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < n[0])
      dest[idx] = (vectNew[idx] - vect[idx]) * (vectNew[idx] - vect[idx]);
}
#endif

#ifdef GPU
float vectorSubtractAndNormalize2(float *devVectNew, float *devVect, int n) {
   //to be parallelized
   float sum = 0;

   //allocate on gpu
   float *d_difference;
   hipMalloc(&d_difference, n * sizeof(float));
   int *d_n;
   hipMalloc(&d_n, sizeof(int));

   //copy to gpu
   hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

   int numBlocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   //launch kenerl
   subtractAndSquare <<<gridDim, blockDim>>> (devVectNew, devVect, d_difference, d_n);

   //copy result from gpu
   float *difference = (float *) malloc(n * sizeof(float));
   hipMemcpy(difference, d_difference, n * sizeof(float), hipMemcpyDeviceToHost);

   //free from gpu
   hipFree(d_difference);
   hipFree(d_n);

   sum = thrust::reduce(difference, difference + n);
/*
   #else //neither mic nor gpu
   for (int i = 0; i < n; i++) {
      sum += (v[i] - last_v[i]) * (v[i] - last_v[i]);
   }
   #endif
*/
   return sqrtf(sum);
}
#endif

void convertCOO2CSR(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowPtr;

   //hipsparseStatus_t status;
   hipMalloc(&devRowPtr, M->nnz * sizeof(int));

   hipsparseXcoo2csr(handle, M->devRowInd, M->nnz, M->width, devRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
   M->devRowPtr = devRowPtr;

   hipFree(M->devRowInd);
}

void putMatOnDevice(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowInd, *devColInd;
   float *devVal;
   hipMalloc(&devRowInd, M->nnz * sizeof(int));
   hipMemcpy(devRowInd, M->cooRowIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 
   hipMalloc(&devVal, M->nnz * sizeof(float));
   hipMemcpy(devVal, M->cooValA, M->nnz * sizeof(float), hipMemcpyHostToDevice);
   hipMalloc(&devColInd, M->nnz * sizeof(int));
   hipMemcpy(devColInd, M->cooColIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 

   M->devRowInd = devRowInd;
   M->devVal = devVal;
   M->devColInd = devColInd;
}

#ifdef GPU
void sparse_MatrixVectorMultiply(SparseMatrix *M, hipsparseHandle_t handle, float *vect, float *newVect, float **devVect, float **devVectNew) {
   float alpha = 1.0f;
   float beta = 0.0f;
   int vectWidth = M->width;

   hipsparseMatDescr_t descr;
   hipsparseCreateMatDescr(&descr);
   hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

   // Check to see if the prestige vector is already on card
   if ( *devVect == NULL ) {
      hipMalloc(devVect, vectWidth * sizeof(float));
      hipMemcpy(*devVect, vect, vectWidth * sizeof(float), hipMemcpyHostToDevice);
   } 
   if ( *devVectNew == NULL ) {
      hipMalloc(devVectNew, vectWidth * sizeof(float));
   }

   hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
   hipsparseScsrmv(handle, op, M->width, M->width, M->nnz, &alpha,
         descr, M->devVal, M->devRowPtr, M->devColInd, *devVect, &beta, *devVectNew); 

   hipMemcpy(newVect, *devVectNew, vectWidth * sizeof(float), hipMemcpyDeviceToHost);
}
#endif

__global__ void MXplusBKernel(int n, float m, float *x, float b) {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) {
      x[i] = m * x[i] + b;
   }
}

void sparse_MXplusB(SparseMatrix *M, float m, float b) {
   int numBlocks = (M->nnz + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   MXplusBKernel<<<gridDim, blockDim>>>(M->nnz, m, M->devVal, b);
}

void pageRank(SparseMatrix *M, int *array) {
   int n = M->width;
   float *vect = (float *) malloc(sizeof(float) * n);
   std::fill(vect, vect + n, 1.0f / n);

   float *newVect = (float *) malloc(sizeof(float) * n);
   float *devVect = NULL;
   float *devVectNew = NULL;

   hipsparseHandle_t handle;
   hipsparseCreate(&handle);

   putMatOnDevice(M, handle);
   sparse_MXplusB(M, DAMPING_FACTOR, (1.0f - DAMPING_FACTOR) / n);
   convertCOO2CSR(M, handle);
   //int iter = 0;

   float error;
   do {
      sparse_MatrixVectorMultiply(M, handle, vect, newVect, &devVect, &devVectNew);

      error = vectorSubtractAndNormalize2(devVectNew, devVect, n);

      // Swap old and new vectors to reuse space
      float *temp = newVect;
      newVect = vect;
      vect = temp;

      temp = devVectNew;
      devVectNew = devVect;
      devVect = temp;
      //printf("Iteration: %d... Error: %.4f\n", iter++, error);
      
   } while (error > QUADRATIC_ERROR);

   hipFree(devVectNew);
   hipFree(M->devVal);
   hipFree(M->devRowPtr);
   hipFree(M->devColInd);
   
   free(newVect);

   for(int i = 0; i < M->width; i++) {
      array[i] = i;
   }

   thrust::stable_sort_by_key(vect, vect + M->width, array, thrust::greater<float>());

   M->sortedPrestigeVector = vect;
}
/*
int main() {
   float vals[7] = {.5, .5, .5, .5, .5, 1, .5};
   int rowInd[7] = {0, 0, 1, 1, 2, 2, 3};
   int colInd[7] = {0, 1, 1, 2, 0, 3, 2};

   float vals2[9] = {1.0, .25, 1, 1, .5, .25, .25, .25, .5};
   int rowInd2[9] = {0, 1, 1, 1, 1, 2, 3, 4, 4};
   int colInd2[9] = {1, 0, 2, 3, 4, 0, 0, 0, 4};

   SparseMatrix m (vals, rowInd, colInd, 4, 7);
   SparseMatrix m2(vals2, rowInd2, colInd2, 5, 9);

   pageRank(m);
   pageRank(m2);
     
   return 0;
}*/
