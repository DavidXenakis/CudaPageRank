#include "hip/hip_runtime.h"
/**
 * Implementation by David Xenakis and Max Bendick
 */
#include "SparseMatrix.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include "hipsparse.h"

#define QUADRATIC_ERROR .001
#define DAMPING_FACTOR .85
#define THREADS_PER_BLOCK 1024

// This function does vector subtraction and absolute value
__global__ void subtractAndSquare(float *vectNew, float *vect, float *dest, int n) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   float val;
   if (idx < n){
      val = (vectNew[idx] - vect[idx]);
      dest[idx] = val < 0 ? val * -1.0 : val;
   }
}

// This function finds the total error across iterations
float vectorSubtractAndNormalize2(float *devVectNew, float *devVect, float *devDifference, float *difference, int n) {
   //to be parallelized
   float sum = 0;

   //allocate on gpu

   int numBlocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   //launch kenerl
   subtractAndSquare <<<gridDim, blockDim>>> (devVectNew, devVect, devDifference, n);

   //copy result from gpu
   hipMemcpy(difference, devDifference, n * sizeof(float), hipMemcpyDeviceToHost);

   sum = thrust::reduce(difference, difference + n);

   return sum;
}

// This function converts our COO sparse matrix into CSR format
void convertCOO2CSR(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowPtr;

   //hipsparseStatus_t status;
   hipMalloc(&devRowPtr, M->nnz * sizeof(int));

   hipsparseXcoo2csr(handle, M->devRowInd, M->nnz, M->width, devRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
   M->devRowPtr = devRowPtr;

   hipFree(M->devRowInd);
}

void putMatOnDevice(SparseMatrix *M, hipsparseHandle_t handle) {
   int *devRowInd, *devColInd;
   float *devVal;

   hipMalloc(&devRowInd, M->nnz * sizeof(int));
   hipMemcpy(devRowInd, M->cooRowIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 
   hipMalloc(&devVal, M->nnz * sizeof(float));
   hipMemcpy(devVal, M->cooValA, M->nnz * sizeof(float), hipMemcpyHostToDevice);
   hipMalloc(&devColInd, M->nnz * sizeof(int));
   hipMemcpy(devColInd, M->cooColIndA, M->nnz * sizeof(int), hipMemcpyHostToDevice); 

   M->devRowInd = devRowInd;
   M->devVal = devVal;
   M->devColInd = devColInd;
}

// This function finds the new prestige vector by multiplying the matrix times the prestige vector
void sparse_MatrixVectorMultiply(SparseMatrix *M, hipsparseHandle_t handle, float *devVect, float *devVectNew) {
   float alpha = 1.0f;
   float beta = 0.0f;

   hipsparseMatDescr_t descr;
   hipsparseCreateMatDescr(&descr);
   hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

   hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
   hipsparseScsrmv(handle, op, M->width, M->width, M->nnz, &alpha,
         descr, M->devVal, M->devRowPtr, M->devColInd, devVect, &beta, devVectNew); 
}

// This function is used to initialize the Matrix with dampening
__global__ void MXKernel(int n, float m, float *x) {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) {
      x[i] = m * x[i];
   }
}

// This function is used to normalize the prestige vectors
__global__ void XplusBKernel(int n, float *x, float b) {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if (i < n) {
      x[i] = x[i] + b;
   }
}

// This function is used to initialize the Matrix with dampening
void sparse_MX(SparseMatrix *M, float m) {
   int numBlocks = (M->nnz + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   MXKernel<<<gridDim, blockDim>>>(M->nnz, m, M->devVal);
}

// This function is used to normalize the prestige vectors
void sparse_XplusB(float *devVect, int width, float b) {
   int numBlocks = (width + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

   dim3 gridDim(numBlocks, 1);
   dim3 blockDim(THREADS_PER_BLOCK, 1);

   XplusBKernel<<<gridDim, blockDim>>>(width, devVect, b);
}

// This function takes a sparse matrix and does the page rank algorithm.
// It returns a sorted list of prestige
void pageRank(SparseMatrix *M, int *array) {
   int n = M->width;
   printf("Width: %d\n", n);
   float *vect = (float *) malloc(sizeof(float) * n);
   std::fill(vect, vect + n, 1.0f / n);
   float *difference = (float *) malloc(n * sizeof(float));
   float *devDifference = NULL;
   float *devVect = NULL;
   float *devVectNew = NULL;

   // Create handle to reuse data on GPU
   hipsparseHandle_t handle;
   hipsparseCreate(&handle);

   // Initialize value for normalizing prestige vectors 
   float b = (1.0f - DAMPING_FACTOR) / n;

   //Send all matrix data to card
   putMatOnDevice(M, handle);

   // Initialize Matrix with dampening
   sparse_MX(M, DAMPING_FACTOR);

   //Convert COO matrix to CSR format
   convertCOO2CSR(M, handle);


   hipMalloc(&devDifference, n * sizeof(float));
   hipMalloc(&devVect, n * sizeof(float));
   hipMalloc(&devVectNew, n * sizeof(float));
   hipMemcpy(devVect, vect, n * sizeof(float), hipMemcpyHostToDevice);
   
   int iter = 0;
   float error;

   do {
      // Find new Prestige vector
      sparse_MatrixVectorMultiply(M, handle, devVect, devVectNew);

      // normalize prestige vector
      sparse_XplusB(devVectNew, n, b);

      // Find error
      error = vectorSubtractAndNormalize2(devVectNew, devVect, devDifference, difference, n);

      //Swap pointers for reuse
      float *temp = devVectNew;
      devVectNew = devVect;
      devVect = temp;
      printf("Iteration: %d... Error: %.7f\n", iter++, error);
      
   } while (error > QUADRATIC_ERROR);

   hipMemcpy(vect, devVect, n * sizeof(float), hipMemcpyDeviceToHost);

   hipFree(devVectNew);
   hipFree(devVect);
   hipFree(M->devVal);
   hipFree(M->devRowPtr);
   hipFree(M->devColInd);
   hipFree(devDifference);

   free(difference); 
   //free(newVect);

   for(int i = 0; i < M->width; i++) {
      array[i] = i;
   }

   // Sort prestige vector
   thrust::stable_sort_by_key(vect, vect + n, array, thrust::greater<float>());

   M->sortedPrestigeVector = vect;
}
